#include "hip/hip_runtime.h"
// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009
//
//   Ulm University
//
// Creator: Hendrik Lensch, Holger Dammertz
// Email:   hendrik.lensch@uni-ulm.de, holger.dammertz@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>

#include "PPM.hh"

using namespace std;
using namespace ppm;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

__device__ __constant__ float3 gpuClusterCol[2048];

#define THREADS 256
#define LOG_IMG_SIZE 8
#define IMG_SIZE 256
#define WINDOW 6

/* The function measures for every pixel the distance to all
 clusters, and determines the clusterID of the nearest cluster
 center. It then colors the pixel in the cluster's color.

 The cluster centers are given as an array of linear indices into
 the vector image, i.e.    _clusterInfo[0] = (x_0 + y_0 * _w).

 */
__global__ void voronoiKernel(float3* _dst, int _w, int _h, int _nClusters, const int* _clusterInfo)
{
    // get the shared memory
    extern __shared__ int shm[];

    int nIter = _nClusters / THREADS + 1;
    // load cluster data
    for (int i = 0; i < nIter; ++i)
    {
        int pos = i * THREADS + threadIdx.x;
        if (pos < _nClusters)
        {
            shm[pos] = _clusterInfo[pos];
        }
    }

    __syncthreads();

    // compute the position within the image
    float x = blockIdx.x * blockDim.x + threadIdx.x;
    float y = blockIdx.y;

    int pos = x + y * _w;

    // determine which is the closest cluster
    float minDist = 1000000.;
    int minIdx = 0;
    for (int i = 0; i < _nClusters; ++i)
    {

        float yy = shm[i] >> LOG_IMG_SIZE;
        float xx = shm[i] % IMG_SIZE;

        float dist = (x - xx) * (x - xx) + (y - yy) * (y - yy);
        if (dist < minDist)
        {
            minDist = dist;
            minIdx = i;
        }
    }

    _dst[pos].x = gpuClusterCol[minIdx].x;
    _dst[pos].y = gpuClusterCol[minIdx].y;
    _dst[pos].z = gpuClusterCol[minIdx].z;

    // mark the center of each cluster
    if (minDist <= 2.)
    {
        _dst[pos].x = 255;
        _dst[pos].y = 0.;
        _dst[pos].z = 0.;
    }
}

__device__ float luminance(const float4& _col)
{
    return 0.299 * _col.x + 0.587 * _col.y + 0.114 * _col.z;
}

/** stores a 1 in _dst if the pixel's luminance is a maximum in the
WINDOW x WINDOW neighborhood
 */
__global__ void featureKernel(int* _dst, hipTextureObject_t texImg, int _w, int _h)
{
    // compute the position within the image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y;

    float lum = luminance(tex2D<float4>(texImg, x, y));

    bool maximum = false;

    if (lum > 20)
    {
        maximum = true;
        for (int v = y - WINDOW; v < y + WINDOW; ++v)
        {
            for (int u = x - WINDOW; u < x + WINDOW; ++u)
            {

                if (lum < luminance(tex2D<float4>(texImg, u, v)))
                {
                    maximum = false;
                }
            }
        }
    }

    if (maximum)
    {
        _dst[x + y * _w] = 1;
    }
    else
    {
        _dst[x + y * _w] = 0;
    }
}

// Chapter 39. Parallel Prefix Sum (Scan) with CUDA
/*
    Compaction kernel works for each block individually.
    It takes the input array g_idata and writes the result to g_odata.
    temp is the shared memory - always works with local thread index
    g_odata and g_idata always works with the global thread index

    In the first case: each block corresponds to one scanline of the image.
    At the end, each scanline of gpuFeatureImg will be partially prefix summed.
*/
// TODO: Create a subfuction on device to call from both kernels
__global__ void reductionKernel1(int *g_odata, int *g_idata) {
    extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;
    int bid = blockIdx.x;

    temp[thid] = g_idata[bid * blockDim.x + thid];

    for(int stride = 1; stride <= blockDim.x / 2; stride <<= 1) {
 		int index = (thid+1) * stride * 2 - 1;
 		if(index < blockDim.x) temp[index] += temp[index-stride];
 		__syncthreads();
    }

    __syncthreads();
    g_odata[bid * blockDim.x + thid] = temp[thid];
}

// only for one block - the last element of each scanline
__global__ void reductionKernel2(int *g_odata, int *g_idata, int w) {
    extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;
    // assign each thread to the last element of a scanline
    int scanLine = w * (thid+1);    // scanline
    int lastElement = scanLine - 1; // last element of scanLine

    temp[thid] = g_idata[lastElement]; // last element of scanLine

    for (int stride = 1; stride <= blockDim.x / 2; stride <<= 1) // build sum in place up the tree
    {
        // here, we add temp[index - stride] to temp[index] if index < blockDim.x
        // simulation of first stride:
        /*
            stride = 1
            0 -> 1    temp[1] += temp[1 - 1]
            1 -> 3    temp[3] += temp[3 - 1]
            2 -> 5    temp[5] += temp[5 - 1]
            3 -> 7    temp[7] += temp[7 - 1]
            4 -> 9   x  (exceeds the boundary)

        */
 		int index = (thid+1) * stride * 2 - 1;
 		if(index < blockDim.x) temp[index] += temp[index-stride];

    }

    __syncthreads();
    g_odata[lastElement] = temp[thid];
}


__global__ void spreadingKernel2(int *g_odata, int *g_idata, int w) {
	extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;
    int bid = blockIdx.x;

    // slighly different from compactionKernel1
    int scanLine = w * (thid);    // scanline
    int lastElement = scanLine - 1; // last element of scanLine

    temp[thid] = g_idata[lastElement]; // last element of scanLine
    if(thid==0) temp[thid] = 0;
    __syncthreads();

    for(int stride=blockDim.x/2; stride > 0; stride>>=1) {
        __syncthreads();
        /**
         * Simulation for example: 25 43 44 123
         * We have 5 threads:  0 25 43 44 123
         *
         * stride = 2
         * thid=0  --  idx=0 --  temp[2] += temp[0]   (new array= 0 25 43 44 123)
         * thid=1  --  idx=4 --  temp[6] out of boundary - stop
         *
         * stride = 1
         * thid=0  --  idx=0 --  temp[1] += temp[0]   (new array= 0 25 43 44 123)
         * thid=1  --  idx=2 --  temp[3] += temp[2]   (new array= 0 25 44 87 123)
         * thid=2  --  idx=4 --  temp[5] out of boundary - stop
         *
        */
        int idx = 2 * stride * thid;
        if(idx + stride < blockDim.x) temp[idx + stride] += temp[idx];
    }
    __syncthreads();
    g_odata[lastElement] = temp[thid];

}


// each block corresponds to one scanline of the image.
__global__ void spreadingKernel1(int *g_odata, int *g_idata) {
    extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;
    int bid = blockIdx.x;

    int scanLineBeginning = bid * (blockDim.x-1);

    if(thid != 0) temp[thid] = g_idata[scanLineBeginning + thid - 1];
    else temp[0] = 0;
    __syncthreads();

    for(int stride=blockDim.x/2; stride > 0; stride>>=1) {
        __syncthreads();
        int idx = 2 * stride * thid;
        if(idx + stride < blockDim.x) temp[idx + stride] += temp[idx];
    }
    __syncthreads();
    if(thid != 0) g_odata[bid * (blockDim.x-1) + thid - 1] = temp[thid];
    // add the previous
    __syncthreads();
    if(thid != blockDim.x - 1 && thid != 0) { // if not last or first (0) element of scanline
    	g_odata[scanLineBeginning + thid - 1] += scanLineBeginning - 1 >= 0 ? g_odata[scanLineBeginning - 1] : 0;
    }
    __syncthreads();
}


__global__ void shiftKernel(int *g_odata, int *g_idata) {
    // copy the input array to the output array with blocks
	int thid = threadIdx.x;
	int x = blockIdx.x * blockDim.x +thid;
    g_odata[x + 1] = g_idata[x];
    if(x == 0) g_odata[x] = 0;
    __syncthreads();
}


// Slides page 37
__global__ void compactKernel(int *g_odata, int *g_idata) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(g_idata[thid] != g_idata[thid + 1]) g_odata[g_idata[thid]] = thid;
}

// !!! missing !!!
// Kernels for Prefix Sum calculation (compaction, spreading, possibly shifting)
// and for generating the gpuFeatureList from the prefix sum.

/* This program detects the local maxima in an image, writes their
location into a vector and then computes the Voronoi diagram of the
image given the detected local maxima as cluster centers.

A Voronoi diagram simply colors every pixel with the color of the
nearest cluster center. */

int main(int argc, char* argv[])
{

    // parse command line
    int acount = 1;
    if (argc < 4)
    {
        printf("usage: testPrefix <inImg> <outImg> <mode>\n");
        exit(1);
    }
    string inName(argv[acount++]);
    string outName(argv[acount++]);
    int mode = atoi(argv[acount++]);

    // Load the input image
    float* cpuImage;
    int w, h;
    readPPM(inName.c_str(), w, h, &cpuImage);
    int nPix = w * h;  // 65536

    // Allocate GPU memory
    int* gpuFeatureImg; // Contains 1 for a feature, 0 else
    // Can be used to do the reduction step of prefix sum calculation in place
    int* gpuPrefixSumShifted; // Output buffer containing the prefix sum
    // Shifted by 1 since it contains 0 as first element by definition
    int* gpuFeatureList; // List of pixel indices where features can be found.
    float3* gpuVoronoiImg; // Final rgb output image
    hipMalloc((void**)&gpuFeatureImg, (nPix) * sizeof(int));

    hipMalloc((void**)&gpuPrefixSumShifted, (nPix + 1) * sizeof(int));
    hipMalloc((void**)&gpuFeatureList, 10000 * sizeof(int));

    hipMalloc((void**)&gpuVoronoiImg, nPix * 3 * sizeof(float));

    // color map for the cluster
    float clusterCol[2048 * 3];
    float* ci = clusterCol;
    for (int i = 0; i < 2048; ++i, ci += 3)
    {
        ci[0] = 32 * i % 256;
        ci[1] = (10 * i + 128) % 256;
        ci[2] = (40 * i + 255) % 256;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(gpuClusterCol), clusterCol, 2048 * 3 * sizeof(float));

    hipArray* gpuTex;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float4>();
    hipMallocArray(&gpuTex, &floatTex, w, h);

    // pad to float4 for faster access
    float* img4 = new float[w * h * 4];

    for (int i = 0; i < w * h; ++i)
    {
        img4[4 * i] = cpuImage[3 * i];
        img4[4 * i + 1] = cpuImage[3 * i + 1];
        img4[4 * i + 2] = cpuImage[3 * i + 2];
        img4[4 * i + 3] = 0.;
    }

    // upload to array

    hipMemcpy2DToArray(gpuTex, 0, 0, img4, w * 4 * sizeof(float), w * 4 * sizeof(float), h,
                        hipMemcpyHostToDevice);

    // create texture object
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = gpuTex;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);

    cout << "setup texture" << endl;
    cout.flush();

    // calculate the block dimensions
    dim3 threadBlock(THREADS);
    dim3 blockGrid(w / THREADS, h, 1);

    printf("blockDim: %d  %d \n", threadBlock.x, threadBlock.y);
    printf("gridDim: %d  %d \n", blockGrid.x, blockGrid.y);

    featureKernel<<<blockGrid, threadBlock>>>(gpuFeatureImg, tex, w, h);

    // variable to store the number of detected features = the number of clusters
    int nFeatures;

    if (mode == 0)
    {
        ////////////////////////////////////////////////////////////
        // CPU compaction:
        ////////////////////////////////////////////////////////////

        // download result

        hipMemcpy(cpuImage, gpuFeatureImg, nPix * sizeof(float), hipMemcpyDeviceToHost);

        std::vector<int> features;

        // add indices of features where gpuFeatureImg != 0
        float* ii = cpuImage;
        for (int i = 0; i < nPix; ++i, ++ii)
        {
            if (*ii > 0)
            {
                features.push_back(i);
            }
        }

        cout << "nFeatures: " << features.size() << endl;

        nFeatures = features.size();
        // upload feature vector

        hipMemcpy(gpuFeatureList, &(features[0]), nFeatures * sizeof(int), hipMemcpyHostToDevice);
    }
    else
    {
        ////////////////////////////////////////////////////////////
        // GPU compaction:
        ////////////////////////////////////////////////////////////



        /*
         gpuFeatureImg -->  nPix many 0s and 1s
         First we want to apply reduction on each scanline:
         there are h many scanlines, so we need h many blocks
        */
        // !!! missing !!!
        // implement the prefixSum algorithm
        // 1. Do the reduction step for all scanlines, one scanline per block.

        reductionKernel1<<<h, w, w * sizeof(int)>>>(gpuFeatureImg, gpuFeatureImg);
        reductionKernel2<<<1, h, h * sizeof(int)>>>(gpuFeatureImg, gpuFeatureImg, w);
        spreadingKernel2<<<1, h+1, (h+1) * sizeof(int)>>>(gpuFeatureImg, gpuFeatureImg, w);
        spreadingKernel1<<<h, (w+1), (w+1) * sizeof(int)>>>(gpuFeatureImg, gpuFeatureImg);
        shiftKernel<<<h, w>>>(gpuPrefixSumShifted, gpuFeatureImg);
        hipMemset(gpuFeatureList, 0, 10000 * sizeof(int));
        compactKernel<<<h, w>>>(gpuFeatureList, gpuPrefixSumShifted);
        // nFeatures is equal to the last element of gpuPrefixSumShifted
        hipMemcpy(&nFeatures, gpuPrefixSumShifted + nPix, sizeof(int), hipMemcpyDeviceToHost);

        // 2. Do the reduction step for the last elements of all scanlines, all in one block

        // 3. Do the spreading step for the last elements of all scanlines, all in one block.
        //    -> The last elements / elements before the scanlines have the right values now.
        // 4. Do the spreading step for all scanlines, one scanline per block.

        // Make sure that gpuFeatureList is filled according to the CPU implementation
        // and that nFeatures has the correct value!


    }

    // now compute the Voronoi Diagram around the detected features.
    voronoiKernel<<<blockGrid, threadBlock, nFeatures * sizeof(int)>>>(gpuVoronoiImg, w, h,
                                                                       nFeatures, gpuFeatureList);

    // download final voronoi image.

    hipMemcpy(cpuImage, gpuVoronoiImg, nPix * 3 * sizeof(float), hipMemcpyDeviceToHost);
    // Write to disk
    writePPM(outName.c_str(), w, h, (float*)cpuImage);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(gpuTex);
    hipFree(gpuFeatureList);
    hipFree(gpuFeatureImg);
    hipFree(gpuPrefixSumShifted);
    hipFree(gpuVoronoiImg);

    delete[] cpuImage;
    delete[] img4;

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

