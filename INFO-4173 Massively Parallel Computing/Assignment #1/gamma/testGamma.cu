#include "hip/hip_runtime.h"
// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009-2010
//
//   Ulm University
//
// Creator: Hendrik Lensch
// Email:   {hendrik.lensch,johannes.hanika}@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include "PPM.hh"

using namespace std;
using namespace ppm;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define MAX_THREADS 128

//-------------------------------------------------------------------------------

// specify the gamma value to be applied
__device__ __constant__ float gpuGamma[1];


/* compute gamma correction on the float image _src of resolution dim,
 outputs the gamma corrected image should be stored in_dst[blockIdx.x *
 blockDim.x + threadIdx.x]. Each thread computes on pixel element.
 */
__global__ void absKernel(float* _dst, const float* _src1, const float* _src2, int _w)
{
    int x = blockIdx.x * MAX_THREADS + threadIdx.x;
    int y = blockIdx.y;
    int pos = y * _w + x;

    if (x < _w)
    {
        _dst[pos] = _src1[pos] - _src2[pos] > 0 ? _src1[pos] - _src2[pos] : _src2[pos] - _src1[pos];
    }
}

//-------------------------------------------------------------------------------

int main(int argc, char* argv[])
{
    int acount = 1; // parse command line

    if (argc < 3)
    {
        printf("usage: %s <inImg> <inImg2>\n", argv[0]);
        exit(1);
    }

    float* img1;
    float* img2;
    float* img3;

    bool success = true;
    int w, h;
    success &= readPPM(argv[acount++], w, h, &img1);
    if (!success) {
        exit(1);
    }
    success &= readPPM(argv[acount++], w, h, &img2);
    if (!success) {
        exit(1);
    }
    int nPix = w * h;

    float* gpuImg1;
    float* gpuImg2;
    float* gpuResImg;

    //-------------------------------------------------------------------------------
    printf("Executing the GPU Version\n");
    // copy the image to the device
    hipHostMalloc((void**)&img3, nPix * 3 * sizeof(float));
    hipMalloc((void**)&gpuImg1, nPix * 3 * sizeof(float));
    hipMalloc((void**)&gpuImg2, nPix * 3 * sizeof(float));
    hipMalloc((void**)&gpuResImg, nPix * 3 * sizeof(float));
    hipMemcpy(gpuImg1, img1, nPix * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpuImg2, img2, nPix * 3 * sizeof(float), hipMemcpyHostToDevice);


    // calculate the block dimensions
    dim3 threadBlock(MAX_THREADS);
    // select the number of blocks vertically (*3 because of RGB)
    dim3 blockGrid((w * 3) / MAX_THREADS + 1, h, 1);
    printf("bl/thr: %d  %d %d\n", blockGrid.x, blockGrid.y, threadBlock.x);

    absKernel<<<blockGrid, threadBlock>>>(gpuResImg, gpuImg1, gpuImg2, w * 3);

    // download result
    hipMemcpy(img3, gpuResImg, nPix * 3 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpuResImg);
    hipFree(gpuImg1);
    hipFree(gpuImg2);

    writePPM(argv[acount++], w, h, (float*)img3);

    delete[] img1;
    delete[] img2;
    delete[] img3;

    checkCUDAError("end of program");

    printf("  done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
